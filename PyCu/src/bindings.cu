#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>


extern "C" {
void *  alloc_gpu_mem( size_t N)
{
	void*d;
	int size = N *sizeof(float);
	int err;

	err = hipMalloc(&d, size);
	if (err != 0) printf("cuda malloc error: %d\n", err);

	return d;
}}


// see kernels.cu for launch_kernel functions

extern "C" {
void host2gpu(float * a, void * da, size_t N)
{
	int size = N * sizeof(float); 
	int err;

	err = hipMemcpy(da, a, size, hipMemcpyHostToDevice);
	if (err != 0) printf("load mem: %d\n", err);

}}
extern "C"{
void gpu2host(float *c, void *d_c, size_t N)
{
	hipError_t  err;
	int size = N*sizeof(float);
	// copy result back
	err = hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	if (err != 0) {printf("cpy mem back %d\n", err);
		//hipError_t hipGetLastError(void);
		printf("%s\n", hipGetErrorString(hipGetLastError()));
	}
}}
extern "C"{
void free_gpu_mem(void *d)
{
	hipFree(d);
}}	

extern "C"{
void free_mem(void *d)
{
	free(d);
}}	


extern "C"{
void get_cuda_info()
{
	int count, i;
	const int kb = 1024;
	const int mb = kb*kb;

	hipGetDeviceCount(&count);
	for(i=0; i<count;i++)
	{
		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, i);
		printf("\nDevice Details:\n");
		printf("%d : %s : %d : %d\n", i, props.name, props.major, props.minor);
		printf("Number of Processors: %d\n", props.multiProcessorCount);
		printf("Global Memory: %f mb\n", (float) props.totalGlobalMem /mb);
		printf("Shared Memory: %f kb \n", (float) props.sharedMemPerBlock / kb);
		printf("Constant Memory: %f kb\n", (float) props.totalConstMem / kb);
		printf("Block registers: %d\n", props.regsPerBlock);

		printf("Warp size: %d\n", props.warpSize);
		printf("Threads per block: %d\n", props.maxThreadsPerBlock);
		printf("Max block dimensions: [%d, %d, %d]\n", props.maxThreadsDim[0], props.maxThreadsDim[1], props.maxThreadsDim[2]);
		printf("Max grid dimensions: [%d, %d, %d]\n", props.maxGridSize[0], props.maxGridSize[1], props.maxGridSize[2]);

		printf("Clock Rate: %d\n", props.memoryClockRate);
		printf("Memory Bus Widths %d\n", props.memoryBusWidth);


		printf("\n");
	}
}}


extern "C"{
void distance3D(float *x, float *y, float *dist, size_t nx, size_t ny, size_t T, size_t k)
{
	int n, m, i, j;
	float d, d_;

	for (n=0; n<nx; n++){
		for (m=0; m<ny; m++){
			d = 0;
			for (i=0; i<T; i++){
				d_ = 0;
				for (j =0; j<k; j++){
					d_ += pow((x[n*T*k + i*k + j] - y[m*T*k + i*k + j]), 2);
				}
				d += sqrt(d_);
			}
			dist[n*ny + m] = d;
		}
	}
}}
