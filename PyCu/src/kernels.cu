
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__device__ float ed1D(float *x, float *y, int K){
	int i;
	float d=0;

	for(i=0;i<K;i++)
		d += pow((x[i] - y[i]), 2);
	
	return sqrt(d);
}

__device__ float ed2D(float *x, float *y, int T, int K){
	int i;
	float d=0;
	
	for (i =0; i < T; i++){	
		d += ed1D(&x[i*K], &y[i*K], K);
	}

	return d;
}

#define R 10
__device__ float DTW(float *x, float *y, int T, int K){
	int i, j;
	float dd[2*R+1][2]; // use this to walk through the dtw matrix one column at a time (only for values within constraint)
	float n1, n2, n3, min; // dtw neighbours

	// first column is just distances (between first R+1 of x-series and first frame of y)	
	dd[0][0] = ed1D(x, y, K);
	for(j=1;j<R+1;j++){
		dd[j][0] = ed1D(&x[j*K], y, K) + dd[j-1][0];
	}

	// now we step across the y-series.  for each frame, we compute the next column of the dtw matrix, alternating its placement in the one of two columns on dd
	
	int v = 1; // the next column of dtw-matrix is written into  this column of dd
	int w = 0;

	for(i=1;i<T;i++){
		// the diagonal of the dtw matrix in (i, j) is (i, 0).  i+j takes values from R below to R above the diagonal (indexing the rows (frames of x-series) in DTW matrix).  j+R indexes dd.
		for(j= -R ;j< R+1 ;j++){
			// bottom row is just the distancess (between first R+1 of y-series and first frame of x)
			if (i+j==0){
				dd[j+R][v] = ed1D(x, &y[i*K], K) + dd[0][w];
			}
			
			else if (i+j > 0 && i+j < T){
				
				n1 = dd[j+R+1][w];
				n2 = dd[j+R][w];
				n3 = dd[j+R-1][v];
				
				min = fminf(n1, n2);
				min = fminf(min, n3);	
				
				dd[j][v] = ed1D(&x[(i+j)*K], &y[i*K], K) + min;
			}
		}
		v = (v+1)%2;
		w = (w+1)%2;
	}
	return dd[2*R][w];
}

/* 	KNN	*/
__global__ void knn(float *train, float *test, float *dist, size_t N_train, size_t T, size_t K, size_t d_mode)
{
	/* each block processes an instance of test (load it into shared)
	   each thread loads a few elements of the test instance into shared memory
	   each thread computes the distance between the test instance and a train instance, until all train instances have been seen.

	   Assumptions:
	     - NUM_THREADS = N_TRAIN (so a single block can process the whole train set).
	*/

	extern __shared__ float Q[]; // query (test instance);

	int NUM_THREADS = N_train;
	int bx = blockIdx.x; // indexes the test instance (we have as many blocks as test instances)
	int tx = threadIdx.x; // indexes the train instance
	int M = T*K; // size of a test instance (to load into shared memory)
	int S = (T + NUM_THREADS-1) / NUM_THREADS; //number of frames in time series loaded by each thread
	int rr = S*K; // num elements each thread must load
	float d;
	int i;

	// load test instance into shared (each thread loads R elements of test instance, corresponding to S rows of Q)
	for (i=0; i < rr; i++){
		int frame_num = tx*S + i/K;
		if (frame_num < T)
			Q[(tx*S + i/K)*K + (i%K)] = test[bx * M + tx * rr + i];
	}
	__syncthreads();	
	
	// compute distance between test instance (Q) and a train instance
	if (d_mode == 0)
		d = ed2D(Q, &train[tx*M], T, K);
	else
		d = DTW(Q, &train[tx*M], T, K);
	
	dist[bx*N_train + tx] = d;

	__syncthreads();
}

extern "C"{
void launch_kernel_knn(void *train, void *test, void *dist, size_t N_train, size_t N_test, size_t T, size_t K, size_t mode)
{
	// for nuw, num_threads and n_train are the same (so we can use a single block to process a test case against the full training set).  Relieve this limitation by looping over the kernel...

	int M = T*K; // for shared memory	
	int NUM_THREADS = N_train;
	knn<<<N_test, NUM_THREADS, M*sizeof(float)>>>((float*)train, (float*)test, (float*)dist, N_train, T, K, mode);

}}

__global__ void dtw_kern(float *Q, float *S, float *D, size_t T, size_t K){

	*D = DTW(Q, S, T, K);
}
extern "C"{
void launch_kernel_dtw(void *Q, void *S, void *D, size_t T, size_t K)
{
	dtw_kern<<<1, 1>>>((float*)Q, (float*)S, (float*)D,T, K);

}}
/* 	Vector Addition */

__global__ void add_0(float *a, float *b, float *c, size_t N){
	int tid = blockIdx.x;
	if (tid < N)
		c[tid] = a[tid] + b[tid];
}

__global__ void add_1(float *a, float *b, float *c, size_t N){
	int tid = threadIdx.x;
	if (tid < N)
		c[tid] = a[tid] + b[tid];
}

__global__ void add_2(float *a, float *b, float *c, size_t N){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < N)
		c[tid] = a[tid] + b[tid];
}


extern "C"{
void launch_kernel_add(void *d_a, void *d_b, void *d_c, size_t N, int mode)
{
	
	// launch kernel
	if (mode == 0)
		add_0<<<N, 1>>>((float*)d_a, (float*)d_b, (float *)d_c, N);
	else if (mode == 1)
		add_1<<<1, N>>>((float*)d_a, (float*)d_b, (float *)d_c, N);
	else
		add_2<<<(N + mode-1)/mode, mode>>>((float *)d_a, (float *)d_b, (float *)d_c, N);
}}


/* Vector Multiplication */
__global__ void multiply_0(float *a, float *b, float *c, size_t N){
	extern __shared__ float temp[];
	int tid = threadIdx.x;
	if (tid < N)
		temp[tid] = a[tid] * b[tid];

	__syncthreads();

	// thread 0 sums pairwise products
	if ( tid == 0){
		float sum = 0;
		for (int i =0; i < N; i++)
			sum += temp[i];
		c[0] = sum;
	}
}

__global__ void multiply_1(float *a, float *b, float *c, size_t N, size_t M){
	c[0] = 0.0; // necessary to initialize since malloc doesn't clear memory
	extern __shared__ float temp[];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < N)
		temp[threadIdx.x] = a[tid] * b[tid];

	__syncthreads();
	
	if (0 == threadIdx.x){
		float sum = 0;
		for (int i=0; i < M; i++)
			sum += temp[i];

		// now we add sum to c.  but since different blocks are doing this potentially simultaneous (read-modify-write), one might read before another writes and hence over write eachother.  so we need to use atomic operations - then read-modify-write is uninteruptable
	atomicAdd(c, sum);
	}
}


extern "C"{
void launch_kernel_multiply(void *d_a, void *d_b, void *d_c, size_t N, size_t M)
{
	printf("%ld, %ld\n", N, M);
	if (M==0)
		multiply_0<<<1, N, N*sizeof(float)>>>((float*)d_a, (float*)d_b, (float *)d_c, N);
	else
		multiply_1<<<(N + M-1)/M, M, M*sizeof(float)>>>((float*)d_a, (float*)d_b, (float *)d_c, N, M);

	printf("%s\n", hipGetErrorString(hipGetLastError()));
}}


/*  Matrix Multiplication */


__global__ void matmult_0(float *a, float *b, float *c, size_t W1, size_t W2, size_t W3){
	float cval = 0;

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	if (row < W1 && col < W3){
		for (int i=0; i<W2; i++)
			cval += a[row*W2 + i] * b[i*W3 + col];
		c[row*W3 + col] = cval;

	}
}

#define TILE_WIDTH 32
__global__ void matmult_1(float *a, float *b, float *c, size_t W1, size_t W2, size_t W3){
	__shared__ float as[TILE_WIDTH][TILE_WIDTH];
	__shared__ float bs[TILE_WIDTH][TILE_WIDTH];

	int bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x, ty = threadIdx.y;
	int row = by * TILE_WIDTH + ty;
	int col = bx * TILE_WIDTH + tx;
	
	float cval =0;

	for (int i =0; i < (W2-1)/TILE_WIDTH+1; ++i){
		if (row < W1 && i*TILE_WIDTH + tx < W2)
			as[ty][tx] = a[row*W2 + i*TILE_WIDTH + tx];
		else
			as[ty][tx] = 0;
		if (col < W3 && i*TILE_WIDTH+ty < W2)
			bs[ty][tx] = b[(i*TILE_WIDTH+ty)*W3 + col];
		else
			bs[ty][tx] = 0;

		__syncthreads();
		for(int k=0; k <TILE_WIDTH; ++k)
			cval += as[ty][k] * bs[k][tx];
		__syncthreads();
	}
	if (row < W1 && col < W3)
		c[row*W3 + col] = cval;

}


extern "C"{
void launch_kernel_matmul(void *da, void *db, void *dc, size_t W1, size_t W2, size_t W3, size_t mode){

	if (mode == 0){
		dim3 dimBlock(16, 16);
		dim3 dimGrid((W3 + dimBlock.x - 1)/dimBlock.x, (W1 + dimBlock.y - 1)/dimBlock.y);
		matmult_0<<<dimGrid, dimBlock>>>((float*)da, (float*)db, (float*)dc, W1, W2, W3);	
	}
	else{
		dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
		dim3 dimGrid((W3-1)/TILE_WIDTH+1, (W1-1)/TILE_WIDTH+1);
		matmult_1<<<dimGrid, dimBlock>>>((float*)da, (float*)db, (float*)dc, W1, W2, W3);

	}

}}














/*
__global__ void stencil_1d(int *in, int *out, int RADIUS){
	__shared__ int temp[BLOCK_SIZE + 2*RADIUS];
	int gindex = threadIdx.x + blockIdx.x*blockDim.x;
	int lindex = threadIdx.x + RADIUS;

	// read input into shared
	temp[lindex] = in[gindex];
	if (threadIdx.x < RADIUS){
		temp[lindex - RADIUS] = in[gindex - RADIUS];
		temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
	}

	__syncthreads();

	int result = 0;
	for (int offset = -RADIUS ; offset <= RADIUS ; offset ++)
		result += temp[lindex + offset];

	out[gindex] = result;
}



*/
